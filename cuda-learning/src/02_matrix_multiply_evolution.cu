#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define TILE_SIZE 16
#define BLOCK_SIZE 16

void check_cuda_error(hipError_t error, const char* msg) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error: %s - %s\n", msg, hipGetErrorString(error));
        exit(1);
    }
}

// 版本1: 朴素的矩阵乘法 - 只使用全局内存
__global__ void matrix_mul_naive(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

// 版本2: 使用共享内存 - 基础tile版本
__global__ void matrix_mul_shared_basic(float* A, float* B, float* C, int N) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    
    float sum = 0.0f;
    
    // 循环处理所有tile
    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // 加载A和B的tile到共享内存
        if (row < N && t * TILE_SIZE + tx < N) {
            As[ty][tx] = A[row * N + t * TILE_SIZE + tx];
        } else {
            As[ty][tx] = 0.0f;
        }
        
        if (col < N && t * TILE_SIZE + ty < N) {
            Bs[ty][tx] = B[(t * TILE_SIZE + ty) * N + col];
        } else {
            Bs[ty][tx] = 0.0f;
        }
        
        __syncthreads();
        
        // 计算当前tile的贡献
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += As[ty][k] * Bs[k][tx];
        }
        
        __syncthreads();
    }
    
    if (row < N && col < N) {
        C[row * N + col] = sum;
    }
}

// 版本3: 优化的共享内存版本 - 避免bank冲突
__global__ void matrix_mul_shared_optimized(float* A, float* B, float* C, int N) {
    // 使用填充避免bank冲突
    __shared__ float As[TILE_SIZE][TILE_SIZE + 1];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE + 1];
    
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    
    float sum = 0.0f;
    
    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // 预取数据到共享内存
        if (row < N && t * TILE_SIZE + tx < N) {
            As[ty][tx] = A[row * N + t * TILE_SIZE + tx];
        } else {
            As[ty][tx] = 0.0f;
        }
        
        if (col < N && t * TILE_SIZE + ty < N) {
            Bs[ty][tx] = B[(t * TILE_SIZE + ty) * N + col];
        } else {
            Bs[ty][tx] = 0.0f;
        }
        
        __syncthreads();
        
        // 使用局部变量减少共享内存访问
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += As[ty][k] * Bs[k][tx];
        }
        
        __syncthreads();
    }
    
    if (row < N && col < N) {
        C[row * N + col] = sum;
    }
}

// 版本4: 进一步优化 - 每个线程计算多个元素
__global__ void matrix_mul_shared_advanced(float* A, float* B, float* C, int N) {
    const int THREAD_TILE_SIZE = 4;  // 每个线程计算4x4个元素
    
    __shared__ float As[TILE_SIZE][TILE_SIZE + 1];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE + 1];
    
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    
    // 每个线程计算多个元素
    float sum[THREAD_TILE_SIZE][THREAD_TILE_SIZE] = {0};
    
    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // 协作加载数据到共享内存
        for (int i = 0; i < THREAD_TILE_SIZE; i++) {
            for (int j = 0; j < THREAD_TILE_SIZE; j++) {
                int row = by * TILE_SIZE + ty * THREAD_TILE_SIZE + i;
                int col_A = t * TILE_SIZE + tx * THREAD_TILE_SIZE + j;
                
                if (row < N && col_A < N && ty * THREAD_TILE_SIZE + i < TILE_SIZE && 
                    tx * THREAD_TILE_SIZE + j < TILE_SIZE) {
                    As[ty * THREAD_TILE_SIZE + i][tx * THREAD_TILE_SIZE + j] = 
                        A[row * N + col_A];
                }
                
                int row_B = t * TILE_SIZE + ty * THREAD_TILE_SIZE + i;
                int col = bx * TILE_SIZE + tx * THREAD_TILE_SIZE + j;
                
                if (row_B < N && col < N && ty * THREAD_TILE_SIZE + i < TILE_SIZE && 
                    tx * THREAD_TILE_SIZE + j < TILE_SIZE) {
                    Bs[ty * THREAD_TILE_SIZE + i][tx * THREAD_TILE_SIZE + j] = 
                        B[row_B * N + col];
                }
            }
        }
        
        __syncthreads();
        
        // 计算多个元素
        for (int i = 0; i < THREAD_TILE_SIZE; i++) {
            for (int j = 0; j < THREAD_TILE_SIZE; j++) {
                for (int k = 0; k < TILE_SIZE; k++) {
                    if (ty * THREAD_TILE_SIZE + i < TILE_SIZE && 
                        tx * THREAD_TILE_SIZE + j < TILE_SIZE) {
                        sum[i][j] += As[ty * THREAD_TILE_SIZE + i][k] * 
                                   Bs[k][tx * THREAD_TILE_SIZE + j];
                    }
                }
            }
        }
        
        __syncthreads();
    }
    
    // 写回结果
    for (int i = 0; i < THREAD_TILE_SIZE; i++) {
        for (int j = 0; j < THREAD_TILE_SIZE; j++) {
            int row = by * TILE_SIZE + ty * THREAD_TILE_SIZE + i;
            int col = bx * TILE_SIZE + tx * THREAD_TILE_SIZE + j;
            
            if (row < N && col < N) {
                C[row * N + col] = sum[i][j];
            }
        }
    }
}

// 验证结果正确性
bool verify_result(float* C_ref, float* C_test, int N) {
    float max_error = 0.0f;
    for (int i = 0; i < N * N; i++) {
        float error = fabs(C_ref[i] - C_test[i]);
        max_error = fmax(max_error, error);
    }
    
    printf("最大误差: %e\n", max_error);
    return max_error < 1e-3;
}

// CPU参考实现
void matrix_mul_cpu(float* A, float* B, float* C, int N) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            float sum = 0.0f;
            for (int k = 0; k < N; k++) {
                sum += A[i * N + k] * B[k * N + j];
            }
            C[i * N + j] = sum;
        }
    }
}

int main() {
    const int N = 1024;  // 矩阵大小
    const int size = N * N * sizeof(float);
    
    printf("矩阵乘法性能对比 (大小: %dx%d)\n", N, N);
    printf("=====================================================\n\n");
    
    // 分配主机内存
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C_ref = (float*)malloc(size);
    float* h_C_test = (float*)malloc(size);
    
    // 初始化矩阵
    srand(42);
    for (int i = 0; i < N * N; i++) {
        h_A[i] = (float)rand() / RAND_MAX;
        h_B[i] = (float)rand() / RAND_MAX;
    }
    
    // CPU参考实现
    printf("计算CPU参考结果...\n");
    auto start_cpu = clock();
    matrix_mul_cpu(h_A, h_B, h_C_ref, N);
    auto end_cpu = clock();
    double cpu_time = ((double)(end_cpu - start_cpu)) / CLOCKS_PER_SEC * 1000;
    
    // 分配设备内存
    float* d_A, *d_B, *d_C;
    check_cuda_error(hipMalloc(&d_A, size), "allocate d_A");
    check_cuda_error(hipMalloc(&d_B, size), "allocate d_B");
    check_cuda_error(hipMalloc(&d_C, size), "allocate d_C");
    
    // 复制数据到设备
    check_cuda_error(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice), "copy A");
    check_cuda_error(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice), "copy B");
    
    // 创建CUDA事件用于计时
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    printf("\nGPU实现对比:\n");
    printf("-----------------------------------------------------\n");
    
    // 测试朴素版本
    hipEventRecord(start);
    matrix_mul_naive<<<grid, block>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float time_naive;
    hipEventElapsedTime(&time_naive, start, stop);
    
    check_cuda_error(hipMemcpy(h_C_test, d_C, size, hipMemcpyDeviceToHost), "copy result");
    bool correct = verify_result(h_C_ref, h_C_test, N);
    
    printf("朴素版本:\n");
    printf("  时间: %.3f ms\n", time_naive);
    printf("  正确性: %s\n", correct ? "✓" : "✗");
    printf("  相比CPU: %.2fx\n\n", cpu_time / time_naive);
    
    // 测试基础共享内存版本
    hipEventRecord(start);
    matrix_mul_shared_basic<<<grid, block>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float time_shared_basic;
    hipEventElapsedTime(&time_shared_basic, start, stop);
    
    check_cuda_error(hipMemcpy(h_C_test, d_C, size, hipMemcpyDeviceToHost), "copy result");
    correct = verify_result(h_C_ref, h_C_test, N);
    
    printf("基础共享内存版本:\n");
    printf("  时间: %.3f ms\n", time_shared_basic);
    printf("  正确性: %s\n", correct ? "✓" : "✗");
    printf("  相比朴素版本: %.2fx\n", time_naive / time_shared_basic);
    printf("  相比CPU: %.2fx\n\n", cpu_time / time_shared_basic);
    
    // 测试优化共享内存版本
    hipEventRecord(start);
    matrix_mul_shared_optimized<<<grid, block>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float time_shared_opt;
    hipEventElapsedTime(&time_shared_opt, start, stop);
    
    check_cuda_error(hipMemcpy(h_C_test, d_C, size, hipMemcpyDeviceToHost), "copy result");
    correct = verify_result(h_C_ref, h_C_test, N);
    
    printf("优化共享内存版本:\n");
    printf("  时间: %.3f ms\n", time_shared_opt);
    printf("  正确性: %s\n", correct ? "✓" : "✗");
    printf("  相比基础版本: %.2fx\n", time_shared_basic / time_shared_opt);
    printf("  相比CPU: %.2fx\n\n", cpu_time / time_shared_opt);
    
    // cuBLAS对比 (如果可用)
    hipblasHandle_t handle;
    if (hipblasCreate(&handle) == HIPBLAS_STATUS_SUCCESS) {
        const float alpha = 1.0f, beta = 0.0f;
        
        hipEventRecord(start);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, 
                   &alpha, d_A, N, d_B, N, &beta, d_C, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        
        float time_cublas;
        hipEventElapsedTime(&time_cublas, start, stop);
        
        printf("cuBLAS (参考):\n");
        printf("  时间: %.3f ms\n", time_cublas);
        printf("  相比优化版本: %.2fx\n", time_shared_opt / time_cublas);
        printf("  相比CPU: %.2fx\n\n", cpu_time / time_cublas);
        
        hipblasDestroy(handle);
    }
    
    printf("性能总结:\n");
    printf("-----------------------------------------------------\n");
    printf("算法                    时间(ms)    相比CPU    相比朴素\n");
    printf("CPU                     %.1f       1.0x       -\n", cpu_time);
    printf("GPU朴素                 %.1f       %.1fx      1.0x\n", 
           time_naive, cpu_time/time_naive);
    printf("GPU共享内存(基础)       %.1f       %.1fx      %.1fx\n", 
           time_shared_basic, cpu_time/time_shared_basic, time_naive/time_shared_basic);
    printf("GPU共享内存(优化)       %.1f       %.1fx      %.1fx\n", 
           time_shared_opt, cpu_time/time_shared_opt, time_naive/time_shared_opt);
    
    // 清理资源
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C_ref);
    free(h_C_test);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}