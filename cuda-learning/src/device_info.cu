#include <stdio.h>
#include <hip/hip_runtime.h>

// 辅助函数声明
int _ConvertSMVer2Cores(int major, int minor);

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        
        printf("Device %d: %s\n", i, prop.name);
        printf("  Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("  SM count: %d\n", prop.multiProcessorCount);
        printf("  CUDA cores: %d\n", _ConvertSMVer2Cores(prop.major, prop.minor) * prop.multiProcessorCount);
        printf("  Global memory: %.2f GB\n", prop.totalGlobalMem/1024.0/1024.0/1024.0);
        printf("  Shared memory per block: %zu KB\n", prop.sharedMemPerBlock/1024);
        printf("  Max threads per block: %d\n", prop.maxThreadsPerBlock);
        printf("  Max threads per SM: %d\n", prop.maxThreadsPerMultiProcessor);
        printf("  Registers per block: %d\n", prop.regsPerBlock);
        printf("  Warp size: %d\n", prop.warpSize);
    }
    return 0;
}

// 辅助函数将SM版本转换为核心数
int _ConvertSMVer2Cores(int major, int minor) {
    typedef struct {
        int SM;
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] = {
        {0x30, 192},
        {0x32, 192},
        {0x35, 192},
        {0x37, 192},
        {0x50, 128},
        {0x52, 128},
        {0x53, 128},
        {0x60,  64},
        {0x61, 128},
        {0x62, 128},
        {0x70,  64},
        {0x72,  64},
        {0x75,  64},
        {0x80,  64},
        {0x86, 128},
        {0x87, 128},
        {0x89, 128},
        {0x90, 128},
        {-1, -1}
    };

    int index = 0;
    while (nGpuArchCoresPerSM[index].SM != -1) {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
            return nGpuArchCoresPerSM[index].Cores;
        }
        index++;
    }
    return nGpuArchCoresPerSM[index-1].Cores;
} 